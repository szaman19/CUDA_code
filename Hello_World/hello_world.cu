
#include <hip/hip_runtime.h>
#include <iostream>

//Keyword __global__ is used to indicate the function will be run on the GPU
__global__ void kernel(int *a, int *b, int *c){
  //This function is compiled by nvcc where as the other functions are handled by g++ or gcc
  *c = *a + *b;
}
int main(int argc, char const *argv[]) {
  /* code */
  int a,b,c = 0;
  int *device_a, *device_b, *device_c;
  int size = sizeof(int);

  //Allocate memory in device/GPU
  hipMalloc((void **)&device_a,size);
  hipMalloc((void **)&device_b,size);
  hipMalloc((void **)&device_c,size);

  a = 2;
  b = 7;
  //Copy memory over to GPU
  hipMemcpy(device_a,&a,size,hipMemcpyHostToDevice);
  hipMemcpy(device_b,&b,size,hipMemcpyHostToDevice);
  //<<<blocknumber, threadnumber>>>
  kernel<<<1,1>>>(device_a,device_b,device_c);
  //Copy back over to CPU
  hipMemcpy(&c,device_c,size,hipMemcpyDeviceToHost);

  std::cout << c << '\n';
  hipFree(device_a);
  hipFree(device_b);
  hipFree(device_c);


  return 0;
}
