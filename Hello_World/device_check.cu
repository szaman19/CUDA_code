
#include <hip/hip_runtime.h>
#include <iostream>
/*
This code is copied/adapted from
https://devblogs.nvidia.com/how-query-device-properties-and-handle-errors-cuda-cc/
*/
using namespace std;
int main(int argc, char const *argv[]) {
  /* code */
  int nDevices;

  hipGetDeviceCount(&nDevices);
  //sets nDevices to the number of CUDA capable devices (GPUs)

  cout << nDevices << '\n';
  // for (int i = 0; i < nDevices; i++) {
  //   cudaDeviceProp prop;
  //   cudaGetDeviceProperties(&prop, i);
  //
  //   cout << "Device Numer: " << i <<"\n";
  //   cout <<"\t"<< "Device Name:"<<prop.name<<"\n";
  //   cout <<"\t"<< "Clock Rate(KHz):"<<prop.memoryClockRate<<"\n";
  //   cout <<"\t"<< "But Width(bits):"<<prop.memoryBusWidth<<"\n";
  //   cout << "\t" << "Memory Bandwidth(GB/s):" <<
  //   2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6 << "\n";
  // }

  return 0;
}
